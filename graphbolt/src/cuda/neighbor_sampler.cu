#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <hiprand/hiprand_kernel.h>
#include <graphbolt/cuda_ops.h>
#include <graphbolt/cuda_sampling_ops.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>

#include <algorithm>
#include <array>
#include <hipcub/hipcub.hpp>
#include <limits>
#include <numeric>
#include <type_traits>

#include "../random.h"
#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

constexpr int BLOCK_SIZE = 128;

/**
 * @brief Fills the random_arr with random numbers and the edge_ids array with
 * original edge ids. When random_arr is sorted along with edge_ids, the first
 * fanout elements of each row gives us the sampled edges.
 */
template <
    typename float_t, typename indptr_t, typename indices_t, typename weights_t,
    typename edge_id_t>
__global__ void _ComputeRandoms(
    const int64_t num_edges, const indptr_t* const sliced_indptr,
    const indptr_t* const sub_indptr, const indices_t* const csr_rows,
    const weights_t* const sliced_weights, const indices_t* const indices,
    const uint64_t random_seed, float_t* random_arr, edge_id_t* edge_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;
  hiprandStatePhilox4_32_10_t rng;
  const auto labor = indices != nullptr;

  if (!labor) {
    hiprand_init(random_seed, i, 0, &rng);
  }

  while (i < num_edges) {
    const auto row_position = csr_rows[i];
    const auto row_offset = i - sub_indptr[row_position];
    const auto in_idx = sliced_indptr[row_position] + row_offset;

    if (labor) {
      constexpr uint64_t kCurandSeed = 999961;
      hiprand_init(kCurandSeed, random_seed, indices[in_idx], &rng);
    }

    const auto rnd = hiprand_uniform(&rng);
    const auto prob =
        sliced_weights ? sliced_weights[i] : static_cast<weights_t>(1);
    const auto exp_rnd = -__logf(rnd);
    const float_t adjusted_rnd = prob > 0
                                     ? static_cast<float_t>(exp_rnd / prob)
                                     : std::numeric_limits<float_t>::infinity();
    random_arr[i] = adjusted_rnd;
    edge_ids[i] = row_offset;

    i += stride;
  }
}

struct IsPositive {
  template <typename probs_t>
  __host__ __device__ auto operator()(probs_t x) {
    return x > 0;
  }
};

template <typename indptr_t>
struct MinInDegreeFanout {
  const indptr_t* in_degree;
  const int64_t* fanouts;
  size_t num_fanouts;
  __host__ __device__ auto operator()(int64_t i) {
    return static_cast<indptr_t>(
        min(static_cast<int64_t>(in_degree[i]), fanouts[i % num_fanouts]));
  }
};

template <typename indptr_t, typename indices_t>
struct IteratorFunc {
  indptr_t* indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) { return indices + indptr[i]; }
};

template <typename indptr_t>
struct AddOffset {
  indptr_t offset;
  template <typename edge_id_t>
  __host__ __device__ indptr_t operator()(edge_id_t x) {
    return x + offset;
  }
};

template <typename indptr_t, typename indices_t>
struct IteratorFuncAddOffset {
  indptr_t* indptr;
  indptr_t* sliced_indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) {
    return thrust::transform_output_iterator{
        indices + indptr[i], AddOffset<indptr_t>{sliced_indptr[i]}};
  }
};

template <typename indptr_t, typename in_degree_iterator_t>
struct SegmentEndFunc {
  indptr_t* indptr;
  in_degree_iterator_t in_degree;
  __host__ __device__ auto operator()(int64_t i) {
    return indptr[i] + in_degree[i];
  }
};

c10::intrusive_ptr<sampling::FusedSampledSubgraph> SampleNeighbors(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes,
    const std::vector<int64_t>& fanouts, bool replace, bool layer,
    bool return_eids, torch::optional<torch::Tensor> type_per_edge,
    torch::optional<torch::Tensor> probs_or_mask) {
  TORCH_CHECK(!replace, "Sampling with replacement is not supported yet!");
  // Assume that indptr, indices, nodes, type_per_edge and probs_or_mask
  // are all resident on the GPU. If not, it is better to first extract them
  // before calling this function.
  auto allocator = cuda::GetAllocator();
  auto num_rows = nodes.size(0);
  auto fanouts_pinned = torch::empty(
      fanouts.size(),
      c10::TensorOptions().dtype(torch::kLong).pinned_memory(true));
  auto fanouts_pinned_ptr = fanouts_pinned.data_ptr<int64_t>();
  for (size_t i = 0; i < fanouts.size(); i++) {
    fanouts_pinned_ptr[i] =
        fanouts[i] >= 0 ? fanouts[i] : std::numeric_limits<int64_t>::max();
  }
  // Finally, copy the adjusted fanout values to the device memory.
  auto fanouts_device = allocator.AllocateStorage<int64_t>(fanouts.size());
  CUDA_CALL(hipMemcpyAsync(
      fanouts_device.get(), fanouts_pinned_ptr,
      sizeof(int64_t) * fanouts.size(), hipMemcpyHostToDevice,
      cuda::GetCurrentStream()));
  auto in_degree_and_sliced_indptr = SliceCSCIndptr(indptr, nodes);
  auto in_degree = std::get<0>(in_degree_and_sliced_indptr);
  auto sliced_indptr = std::get<1>(in_degree_and_sliced_indptr);
  torch::optional<int64_t> num_edges_;
  torch::Tensor sub_indptr;
  torch::optional<torch::Tensor> sliced_probs_or_mask;
  if (probs_or_mask.has_value()) {
    torch::Tensor sliced_probs_or_mask_tensor;
    std::tie(sub_indptr, sliced_probs_or_mask_tensor) = IndexSelectCSCImpl(
        in_degree, sliced_indptr, probs_or_mask.value(), nodes,
        indptr.size(0) - 2, num_edges_);
    sliced_probs_or_mask = sliced_probs_or_mask_tensor;
    num_edges_ = sliced_probs_or_mask_tensor.size(0);
  }
  if (fanouts.size() > 1) {
    torch::Tensor sliced_type_per_edge;
    std::tie(sub_indptr, sliced_type_per_edge) = IndexSelectCSCImpl(
        in_degree, sliced_indptr, type_per_edge.value(), nodes,
        indptr.size(0) - 2, num_edges_);
    std::tie(sub_indptr, in_degree, sliced_indptr) = SliceCSCIndptrHetero(
        sub_indptr, sliced_type_per_edge, sliced_indptr, fanouts.size());
    num_rows = sliced_indptr.size(0);
    num_edges_ = sliced_type_per_edge.size(0);
  }
  // If sub_indptr was not computed in the two code blocks above:
  if (!probs_or_mask.has_value() && fanouts.size() <= 1) {
    sub_indptr = ExclusiveCumSum(in_degree);
  }
  auto max_in_degree = torch::empty(
      1,
      c10::TensorOptions().dtype(in_degree.scalar_type()).pinned_memory(true));
  AT_DISPATCH_INDEX_TYPES(
      indptr.scalar_type(), "SampleNeighborsInDegree", ([&] {
        CUB_CALL(
            DeviceReduce::Max, in_degree.data_ptr<index_t>(),
            max_in_degree.data_ptr<index_t>(), num_rows);
      }));
  auto coo_rows = CSRToCOO(sub_indptr, indices.scalar_type());
  const auto num_edges = coo_rows.size(0);
  const auto random_seed = RandomEngine::ThreadLocal()->RandInt(
      static_cast<int64_t>(0), std::numeric_limits<int64_t>::max());
  auto output_indptr = torch::empty_like(sub_indptr);
  torch::Tensor picked_eids;
  torch::Tensor output_indices;
  torch::optional<torch::Tensor> output_type_per_edge;

  AT_DISPATCH_INDEX_TYPES(
      indptr.scalar_type(), "SampleNeighborsIndptr", ([&] {
        using indptr_t = index_t;
        if (probs_or_mask.has_value()) {  // Count nonzero probs into in_degree.
          GRAPHBOLT_DISPATCH_ALL_TYPES(
              probs_or_mask.value().scalar_type(),
              "SampleNeighborsPositiveProbs", ([&] {
                using probs_t = scalar_t;
                auto is_nonzero = thrust::make_transform_iterator(
                    sliced_probs_or_mask.value().data_ptr<probs_t>(),
                    IsPositive{});
                CUB_CALL(
                    DeviceSegmentedReduce::Sum, is_nonzero,
                    in_degree.data_ptr<indptr_t>(), num_rows,
                    sub_indptr.data_ptr<indptr_t>(),
                    sub_indptr.data_ptr<indptr_t>() + 1);
              }));
        }
        thrust::counting_iterator<int64_t> iota(0);
        auto sampled_degree = thrust::make_transform_iterator(
            iota, MinInDegreeFanout<indptr_t>{
                      in_degree.data_ptr<indptr_t>(), fanouts_device.get(),
                      fanouts.size()});

        // Compute output_indptr.
        CUB_CALL(
            DeviceScan::ExclusiveSum, sampled_degree,
            output_indptr.data_ptr<indptr_t>(), num_rows + 1);

        auto num_sampled_edges =
            cuda::CopyScalar{output_indptr.data_ptr<indptr_t>() + num_rows};

        // Find the smallest integer type to store the edge id offsets.
        // CSRToCOO had synch inside, so it is safe to read max_in_degree now.
        const int num_bits =
            cuda::NumberOfBits(max_in_degree.data_ptr<indptr_t>()[0]);
        std::array<int, 4> type_bits = {8, 16, 32, 64};
        const auto type_index =
            std::lower_bound(type_bits.begin(), type_bits.end(), num_bits) -
            type_bits.begin();
        std::array<torch::ScalarType, 5> types = {
            torch::kByte, torch::kInt16, torch::kInt32, torch::kLong,
            torch::kLong};
        auto edge_id_dtype = types[type_index];
        AT_DISPATCH_INTEGRAL_TYPES(
            edge_id_dtype, "SampleNeighborsEdgeIDs", ([&] {
              using edge_id_t = std::make_unsigned_t<scalar_t>;
              TORCH_CHECK(
                  num_bits <= sizeof(edge_id_t) * 8,
                  "Selected edge_id_t must be capable of storing edge_ids.");
              // Using bfloat16 for random numbers works just as reliably as
              // float32 and provides around %30 percent speedup.
              using rnd_t = hip_bfloat16;
              auto randoms = allocator.AllocateStorage<rnd_t>(num_edges);
              auto randoms_sorted = allocator.AllocateStorage<rnd_t>(num_edges);
              auto edge_id_segments =
                  allocator.AllocateStorage<edge_id_t>(num_edges);
              auto sorted_edge_id_segments =
                  allocator.AllocateStorage<edge_id_t>(num_edges);
              AT_DISPATCH_INDEX_TYPES(
                  indices.scalar_type(), "SampleNeighborsIndices", ([&] {
                    using indices_t = index_t;
                    auto probs_or_mask_scalar_type = torch::kFloat32;
                    if (probs_or_mask.has_value()) {
                      probs_or_mask_scalar_type =
                          probs_or_mask.value().scalar_type();
                    }
                    GRAPHBOLT_DISPATCH_ALL_TYPES(
                        probs_or_mask_scalar_type, "SampleNeighborsProbs",
                        ([&] {
                          using probs_t = scalar_t;
                          probs_t* sliced_probs_ptr = nullptr;
                          if (sliced_probs_or_mask.has_value()) {
                            sliced_probs_ptr = sliced_probs_or_mask.value()
                                                   .data_ptr<probs_t>();
                          }
                          const indices_t* indices_ptr =
                              layer ? indices.data_ptr<indices_t>() : nullptr;
                          const dim3 block(BLOCK_SIZE);
                          const dim3 grid(
                              (num_edges + BLOCK_SIZE - 1) / BLOCK_SIZE);
                          // Compute row and random number pairs.
                          CUDA_KERNEL_CALL(
                              _ComputeRandoms, grid, block, 0, num_edges,
                              sliced_indptr.data_ptr<indptr_t>(),
                              sub_indptr.data_ptr<indptr_t>(),
                              coo_rows.data_ptr<indices_t>(), sliced_probs_ptr,
                              indices_ptr, random_seed, randoms.get(),
                              edge_id_segments.get());
                        }));
                  }));

              // Sort the random numbers along with edge ids, after
              // sorting the first fanout elements of each row will
              // give us the sampled edges.
              CUB_CALL(
                  DeviceSegmentedSort::SortPairs, randoms.get(),
                  randoms_sorted.get(), edge_id_segments.get(),
                  sorted_edge_id_segments.get(), num_edges, num_rows,
                  sub_indptr.data_ptr<indptr_t>(),
                  sub_indptr.data_ptr<indptr_t>() + 1);

              picked_eids = torch::empty(
                  static_cast<indptr_t>(num_sampled_edges),
                  nodes.options().dtype(indptr.scalar_type()));

              // Need to sort the sampled edges only when fanouts.size() == 1
              // since multiple fanout sampling case is automatically going to
              // be sorted.
              if (type_per_edge && fanouts.size() == 1) {
                // Ensuring sort result still ends up in sorted_edge_id_segments
                std::swap(edge_id_segments, sorted_edge_id_segments);
                auto sampled_segment_end_it = thrust::make_transform_iterator(
                    iota, SegmentEndFunc<indptr_t, decltype(sampled_degree)>{
                              sub_indptr.data_ptr<indptr_t>(), sampled_degree});
                CUB_CALL(
                    DeviceSegmentedSort::SortKeys, edge_id_segments.get(),
                    sorted_edge_id_segments.get(), picked_eids.size(0),
                    num_rows, sub_indptr.data_ptr<indptr_t>(),
                    sampled_segment_end_it);
              }

              auto input_buffer_it = thrust::make_transform_iterator(
                  iota, IteratorFunc<indptr_t, edge_id_t>{
                            sub_indptr.data_ptr<indptr_t>(),
                            sorted_edge_id_segments.get()});
              auto output_buffer_it = thrust::make_transform_iterator(
                  iota, IteratorFuncAddOffset<indptr_t, indptr_t>{
                            output_indptr.data_ptr<indptr_t>(),
                            sliced_indptr.data_ptr<indptr_t>(),
                            picked_eids.data_ptr<indptr_t>()});
              constexpr int64_t max_copy_at_once =
                  std::numeric_limits<int32_t>::max();

              // Copy the sampled edge ids into picked_eids tensor.
              for (int64_t i = 0; i < num_rows; i += max_copy_at_once) {
                CUB_CALL(
                    DeviceCopy::Batched, input_buffer_it + i,
                    output_buffer_it + i, sampled_degree + i,
                    std::min(num_rows - i, max_copy_at_once));
              }
            }));

        output_indices = torch::empty(
            picked_eids.size(0),
            picked_eids.options().dtype(indices.scalar_type()));

        // Compute: output_indices = indices.gather(0, picked_eids);
        AT_DISPATCH_INDEX_TYPES(
            indices.scalar_type(), "SampleNeighborsOutputIndices", ([&] {
              using indices_t = index_t;
              THRUST_CALL(
                  gather, picked_eids.data_ptr<indptr_t>(),
                  picked_eids.data_ptr<indptr_t>() + picked_eids.size(0),
                  indices.data_ptr<indices_t>(),
                  output_indices.data_ptr<indices_t>());
            }));

        if (type_per_edge) {
          // output_type_per_edge = type_per_edge.gather(0, picked_eids);
          // The commented out torch equivalent above does not work when
          // type_per_edge is on pinned memory. That is why, we have to
          // reimplement it, similar to the indices gather operation above.
          auto types = type_per_edge.value();
          output_type_per_edge = torch::empty(
              picked_eids.size(0),
              picked_eids.options().dtype(types.scalar_type()));
          AT_DISPATCH_INTEGRAL_TYPES(
              types.scalar_type(), "SampleNeighborsOutputTypePerEdge", ([&] {
                THRUST_CALL(
                    gather, picked_eids.data_ptr<indptr_t>(),
                    picked_eids.data_ptr<indptr_t>() + picked_eids.size(0),
                    types.data_ptr<scalar_t>(),
                    output_type_per_edge.value().data_ptr<scalar_t>());
              }));
        }
      }));

  // Convert output_indptr back to homo by discarding intermediate offsets.
  output_indptr =
      output_indptr.slice(0, 0, output_indptr.size(0), fanouts.size());
  torch::optional<torch::Tensor> subgraph_reverse_edge_ids = torch::nullopt;
  if (return_eids) subgraph_reverse_edge_ids = std::move(picked_eids);

  return c10::make_intrusive<sampling::FusedSampledSubgraph>(
      output_indptr, output_indices, nodes, torch::nullopt,
      subgraph_reverse_edge_ids, output_type_per_edge);
}

}  //  namespace ops
}  //  namespace graphbolt
